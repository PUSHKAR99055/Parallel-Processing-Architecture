
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

#define MATRIX_SIZE 1024
#define BLOCK_DIM 32                
#define TILE_SZE BLOCK_DIM          //Tile size is same as block dimension. defined for better code understandability
#define TEST 0

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if(error != hipSuccess)\
    {\
        std::cout<<"Error: "<<__FILE__<<":"<<__LINE__<<std::endl;\
        std::cout<<"Code: "<<error<<", reason: "<<hipGetErrorString(error)<<std::endl;\
        exit(1);\
    }\
}

typedef struct
{
    float value;
    int16_t row, col;

} matElement;

typedef struct
{
    float value;
    int pathIndex;
} pathElement;

// #endif

void matrixInit(float *a, float *b, float *c)
{
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {   
            a[i * MATRIX_SIZE + j] = rand() / (float)1147654321;
            b[i * MATRIX_SIZE + j] = rand() / (float)1147654321;
            c[i * MATRIX_SIZE + j] = (float)0;
        }
    }
}

__device__ void warpReduce(volatile matElement *newSharedB, int threadId)
{
    if(newSharedB[threadId].value > newSharedB[threadId + 32].value){
        newSharedB[threadId].value = newSharedB[threadId + 32].value;
        newSharedB[threadId].row = newSharedB[threadId + 32].row;
        newSharedB[threadId].col = newSharedB[threadId + 32].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 16].value){
        newSharedB[threadId].value = newSharedB[threadId + 16].value;
        newSharedB[threadId].row = newSharedB[threadId + 16].row;
        newSharedB[threadId].col = newSharedB[threadId + 16].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 8].value){
        newSharedB[threadId].value = newSharedB[threadId + 8].value;
        newSharedB[threadId].row = newSharedB[threadId + 8].row;
        newSharedB[threadId].col = newSharedB[threadId + 8].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 4].value){
        newSharedB[threadId].value = newSharedB[threadId + 4].value;
        newSharedB[threadId].row = newSharedB[threadId + 4].row;
        newSharedB[threadId].col = newSharedB[threadId + 4].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 2].value){
        newSharedB[threadId].value = newSharedB[threadId + 2].value;
        newSharedB[threadId].row = newSharedB[threadId + 2].row;
        newSharedB[threadId].col = newSharedB[threadId + 2].col;
    }

    if(newSharedB[threadId].value > newSharedB[threadId + 1].value){
        newSharedB[threadId].value = newSharedB[threadId + 1].value;
        newSharedB[threadId].row = newSharedB[threadId + 1].row;
        newSharedB[threadId].col = newSharedB[threadId + 1].col;
    }
}

__device__ void minBlockReduce(matElement *newSharedB, int threadId)
{
    for (unsigned int stride = (BLOCK_DIM * BLOCK_DIM)/2; stride > 32; stride >>= 1)
    {
        if(threadId < stride)
        {
            if(newSharedB[threadId].value > newSharedB[threadId + stride].value){
                newSharedB[threadId] = newSharedB[threadId + stride];
            }
        }
        __syncthreads();
    }
    if(threadId < 32) warpReduce(newSharedB, threadId);
}

__global__ void find2Min(int16_t firstMinRow, int16_t firstMinCol, float *c, matElement *d_minValueFromEachBlock)
{
    int16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    int16_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int16_t threadId = threadIdx.y * BLOCK_DIM + threadIdx.x;                       //thread id within each block only

    __shared__ matElement sharedC[BLOCK_DIM * BLOCK_DIM];

    if(row == 0 && col == 0) c[firstMinRow * MATRIX_SIZE + firstMinCol] = __FLT_MAX__;
    
    __syncthreads();

    sharedC[threadId].value = c[row * MATRIX_SIZE + col];
    sharedC[threadId].row = row;
    sharedC[threadId].col = col;
    __syncthreads();

    minBlockReduce(sharedC, threadId);
    if(threadId == 0){   
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].value = sharedC[0].value;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].row = sharedC[0].row;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].col = sharedC[0].col;
    }

    // if(row == 0 && col ==0) c[firstMinRow * MATRIX_SIZE + firstMinCol] = tempVal;                   //replace the first min val with the original since we replaced it with FLT_MAX for finding second min
}

__global__ void tiledMatrixMultiply(float *a, float *b, float *c, matElement *d_minValueFromEachBlock)
{
    int16_t row = blockIdx.y * blockDim.y + threadIdx.y;
    int16_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int16_t threadId = threadIdx.y * BLOCK_DIM + threadIdx.x;

    __shared__ float sharedA[BLOCK_DIM * BLOCK_DIM];
    __shared__ float sharedB[BLOCK_DIM * BLOCK_DIM * sizeof(matElement)];                    

    float temp = 0;

    for (int i = 0; i < MATRIX_SIZE / TILE_SZE; i++)
    {
        sharedA[threadId] = a[row * MATRIX_SIZE + (i * TILE_SZE + threadIdx.x)];                 //index into the global a with the global row (since we are tiling across x dimention of a) and each thread's tile 
        sharedB[threadId] = b[(i * TILE_SZE + threadIdx.y) * MATRIX_SIZE + col];                 //index into the global b with each thread's tile idexes (since we are tiling across y dimention of b) and globale column 
        __syncthreads();                                                                         //make sure all values of the sub-matrices are loaded by thre threads before proceding

        for (int j = 0; j < TILE_SZE; j++)
        {
            temp += sharedA[threadIdx.y * TILE_SZE + j] * sharedB[j * TILE_SZE + threadIdx.x];
        }

        __syncthreads();                                                                         //make sure all sub-matrix calculation is done by threads before advancing to the next sub-matricies

    }
    matElement *newSharedB = (matElement*) sharedB;                                              //reuse shared mem for finding min element

    newSharedB[threadId].value = temp;
    newSharedB[threadId].row = row;
    newSharedB[threadId].col = col;
    __syncthreads();
    
    c[row * MATRIX_SIZE + col] = temp;

    minBlockReduce(newSharedB, threadId);
    if(threadId == 0){   
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].value = newSharedB[0].value;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].row = newSharedB[0].row;
        d_minValueFromEachBlock[blockIdx.y * gridDim.x + blockIdx.x].col = newSharedB[0].col;
    }
}

extern float* computeMatrixMult(matElement *minElement)
{
    struct timeval start_time, end_time;
    double exec_time;
    minElement[0].value = __FLT_MAX__;
    minElement[1].value = __FLT_MAX__;

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    matElement *h_minValueFromEachBlock;
    matElement *d_minValueFromEachBlock;

    size_t size = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);

    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    h_minValueFromEachBlock = (matElement*)malloc((MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement));

    CHECK(hipHostMalloc(&d_a, size, hipHostMallocDefault));
    CHECK(hipHostMalloc(&d_b, size, hipHostMallocDefault));
    CHECK(hipHostMalloc(&d_c, size, hipHostMallocDefault));
    CHECK(hipHostMalloc(&d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement), hipHostMallocDefault));

    matrixInit(h_a, h_b, h_c);

    dim3 blockPerGrid(MATRIX_SIZE / BLOCK_DIM , MATRIX_SIZE / BLOCK_DIM);
    dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);

    
    gettimeofday(&start_time, NULL);

    CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));
    

    tiledMatrixMultiply<<<blockPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, d_minValueFromEachBlock);

    CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_minValueFromEachBlock, d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement), hipMemcpyDeviceToHost));
    for (int i = 0; i < (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM); i++)
    {
        if(h_minValueFromEachBlock[i].value < minElement[0].value)
        {
            minElement[0].value = h_minValueFromEachBlock[i].value;           
            minElement[0].row = h_minValueFromEachBlock[i].row;
            minElement[0].col = h_minValueFromEachBlock[i].col;
        }
    }

    find2Min<<<blockPerGrid, threadsPerBlock>>>(minElement[0].row, minElement[0].col, d_c, d_minValueFromEachBlock);

    CHECK(hipMemcpy(h_minValueFromEachBlock, d_minValueFromEachBlock, (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM) * sizeof(matElement), hipMemcpyDeviceToHost));
    
    d_c[minElement[0].row * MATRIX_SIZE + minElement[0].col] = minElement[0].value;
    
    for (int i = 0; i < (MATRIX_SIZE / BLOCK_DIM) * (MATRIX_SIZE / BLOCK_DIM); i++)
    {
        if(h_minValueFromEachBlock[i].value < minElement[1].value)
        {
            minElement[1].value = h_minValueFromEachBlock[i].value;           
            minElement[1].row = h_minValueFromEachBlock[i].row;
            minElement[1].col = h_minValueFromEachBlock[i].col;
        }
    }
    gettimeofday(&end_time, NULL);

    free(h_a);
    free(h_b);

    hipFree(d_a);
    hipFree(d_b);

    exec_time = (double)(end_time.tv_sec - start_time.tv_sec) + (double)(end_time.tv_usec - start_time.tv_usec)/(double)1000000;

    std::cout<<"Execution time - "<<exec_time<<std::endl;
    
    std::cout<<"Matrix size - "<<MATRIX_SIZE<<std::endl;

    std::cout<<"Min value 1 (val, row, col) - ("<<minElement[0].value<<", "<<minElement[0].row<<", "<<minElement[0].col<<")"<<std::endl;

    std::cout<<"Min value 2 (val, row, col) - ("<<minElement[1].value<<", "<<minElement[1].row<<", "<<minElement[1].col<<")"<<std::endl;

    return d_c;

}

float* computeMatrixMult(matElement*);

void setUpArrays(float *d_c, int *vertex, int *edges, bool *threadMask, float* cost, pathElement* intermediateCost, int* path, matElement* minElement)
{   
    int edgeIndex = 0;
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {   
            threadMask[i * MATRIX_SIZE + j] = false;
            cost[i * MATRIX_SIZE + j] = __FLT_MAX__;
            intermediateCost[i * MATRIX_SIZE + j].value = __FLT_MAX__;
            intermediateCost[i * MATRIX_SIZE + j].pathIndex = -1;
            path[i * MATRIX_SIZE + j] = -1;

            vertex[i * MATRIX_SIZE + j] = edgeIndex;
            if((j + 1) < MATRIX_SIZE) edges[edgeIndex++] = i * MATRIX_SIZE + (j + 1);
        
            if((i + 1) < MATRIX_SIZE) edges[edgeIndex++] = (i + 1) * MATRIX_SIZE + j;

            if((j - 1) >= 0) edges[edgeIndex++] = i * MATRIX_SIZE + (j - 1);

            if((i - 1) >= 0) edges[edgeIndex++] = (i - 1) * MATRIX_SIZE + j;

        }

        threadMask[minElement[0].row * MATRIX_SIZE + minElement[0].col] = true;             //Make the thread of source vertex executable initially since that is the starting point
        cost[minElement[0].row * MATRIX_SIZE + minElement[0].col] = 0.0f;                   //Cost from source to source is 0
        intermediateCost[minElement[0].row * MATRIX_SIZE + minElement[0].col].value = 0.0f;       
    }    
}

void printNeighbors(int index, float *d_c, int *vertex, int* edges)
{
    for (int i = vertex[index]; i < vertex[index + 1]; i++)
    {
        std::cout<<i<<std::endl;
        std::cout<<d_c[edges[i]]<<std::endl;
        std::cout<<"\n";
    }    
}

void printPath(pathElement *path)
{
    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {
            printf("(%d, %d),  ", i * MATRIX_SIZE + j, path[i * MATRIX_SIZE + j].pathIndex);
        }  
    }
}

void findPathBoundaries(pathElement *path)
{

}

/*
    function:   reinterpret the 8 bytes pathElement(4 int + 4 float) into a value of 
                type unsigned long long int which is also of 8 bytes
*/
__device__ unsigned long long int __pathElement_as_ulli(pathElement *pathElement)
{
    unsigned long long int *ulli = reinterpret_cast<unsigned long long int*>(pathElement);
    return *ulli;

}


/*
    function:   reinterpret the 8 bytes unsigned long long int back into a value of 
                type pathElement(4 int + 4 float) which is also of 8 bytes
*/
__device__ pathElement* __ulli_as_pathElement(unsigned long long int *ulli)
{
    pathElement *element = reinterpret_cast<pathElement*>(ulli);
    return element;
}


__device__ __forceinline__ pathElement* atomicMin(pathElement *addr, pathElement* pathElement)
{
    unsigned long long int currentPathElement = __pathElement_as_ulli(addr);                                            //reinterpret to unsigned long long int since atomicCAS() supports it and a few others only
    while (pathElement->value < __ulli_as_pathElement(&currentPathElement)->value)                                         
    {
        unsigned long long int old = currentPathElement;
        currentPathElement = atomicCAS((unsigned long long int*)addr, old, __pathElement_as_ulli(pathElement));          //do atomicCAS on the reinterpreted value of ulli, if *addr == old then it puts value into addr and returns old else it does nothing and just retunrs whatever was there in addr
        if(currentPathElement == old) break;                                                                             //if value was successfully put into addr then the current thread was successful in it's atomic operation else it has to re-run with the new "current value" from addr(that might have been changed by another thread's atomic operation) and do the swapping again
    }
    return __ulli_as_pathElement(&currentPathElement);
}


__global__ void computeIntermediatesAndPath(float *d_c, int *vertex, int *edges, bool *threadMask, float *cost, pathElement *intermediateCost)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int threadIndex = row * MATRIX_SIZE + col;                                              //this is the global thread index to index into the entire matrix and not just for threads within each block
    if(threadMask[threadIndex])
    {
        threadMask[threadIndex] = false;
        for (int i = vertex[threadIndex]; i < vertex[threadIndex + 1]; i++)
        {   
            pathElement costPlusWeightOfCurrentThread;                                      //package cost + weight into pathElement for atomicCAS
            costPlusWeightOfCurrentThread.value = cost[threadIndex] + d_c[edges[i]];
            costPlusWeightOfCurrentThread.pathIndex = threadIndex;
            atomicMin(&intermediateCost[edges[i]], &costPlusWeightOfCurrentThread);
        }
    }
}

__global__ void computeFinalCosts(bool *d_done, int *vertex, int *edges, bool *threadMask, float *cost, pathElement *intermediateCost)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int threadIndex = row * MATRIX_SIZE + col;
    if(intermediateCost[threadIndex].value < cost[threadIndex])
    {
        cost[threadIndex] = intermediateCost[threadIndex].value;
        threadMask[threadIndex] = true;                                                     //since cost of this vertex changed, make it executable again to update it's neigbours
        *d_done = false;                                                                    //no atomicity required as all threads write false value only
    }
    intermediateCost[threadIndex].value = cost[threadIndex];
}

int main()
{

    int *vertex, *edges, *path;
    float *d_c;
    float *cost;
    pathElement *intermediateCostAndPath;
    bool *threadMask;

    bool h_done = false;
    bool *d_done_ptr;

    matElement minElement[2];
    size_t size = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);
    //Note: Diagnonal neighbours are not considered
    int numEdges = (4 * 2                                       /*Each corner values in matrix has 2 neighbours*/ 
                    + ((MATRIX_SIZE - 2) * 3) * 4               /*Each element of the 4 boundary sides excluding the 2 corner elements for each boundary side has 3 neighbours*/ 
                    + (MATRIX_SIZE - 2) * 4 * (MATRIX_SIZE - 2) /*Each element not on the boundary has 4 neighbours*/);
    
    //Compute and get the pointer to the result matrix of the matrix muliplications
    d_c = computeMatrixMult(minElement);

    //Use test data for sssp checking
    #if(TEST)
    float test_data[16] = {1.2, 5.4, 1.0f, 1.0f, 9.7, 4.9, 1.0f, 7.6, 4.0, 8.4, 1.0f, 11.5, 14.3, 2, 30.0f, 17.7};
    CHECK(cudaMemcpy(d_c, &test_data, size, cudaMemcpyHostToDevice));

    for (int i = 0; i < MATRIX_SIZE; i++)
    {
        for (int j = 0; j < MATRIX_SIZE; j++)
        {
            std::cout<<d_c[i * MATRIX_SIZE + j]<<"  ";
        }
        printf("\n");
        
    }
    //Test values for source and target
    minElement[0].row = 3; minElement[0].col = 1;
    minElement[1].row = 3; minElement[1].col = 3;
    #endif   


    //Setup CUDA device memories for the data
    CHECK(hipHostMalloc(&vertex, ((MATRIX_SIZE * MATRIX_SIZE) + 1) * sizeof(int), hipHostMallocDefault));                    // + 1 because we need a location at the end of the vertex that stores the ending index of the edge
    CHECK(hipHostMalloc(&edges, numEdges * sizeof(int), hipHostMallocDefault));
    CHECK(hipHostMalloc(&threadMask, MATRIX_SIZE * MATRIX_SIZE * sizeof(bool), hipHostMallocDefault));
    CHECK(hipHostMalloc(&cost, size, hipHostMallocDefault));
    CHECK(hipHostMalloc(&intermediateCostAndPath, MATRIX_SIZE * MATRIX_SIZE * sizeof(pathElement), hipHostMallocDefault));          //each neighbor need not have it's own cost location because the intermediate cost for a vertex is the same memory location updated by all neighbouring threads.
    CHECK(hipHostMalloc(&path, size, hipHostMallocDefault));
    CHECK(hipHostMalloc(&d_done_ptr, sizeof(bool), hipHostMallocDefault));

    setUpArrays(d_c, vertex, edges, threadMask, cost, intermediateCostAndPath, path, minElement);
    vertex[MATRIX_SIZE * MATRIX_SIZE] = numEdges;                                                       //last value in vertex is total numEdges so that we can use the starting and ending index when getting the neighbors


    dim3 blockPerGrid(MATRIX_SIZE / BLOCK_DIM , MATRIX_SIZE / BLOCK_DIM);
    dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);

    //Start computing SSSP
    while(!h_done)
    {
        h_done = true;

        //memcpy h_done to d_done
        CHECK(hipMemcpy(d_done_ptr, &h_done, sizeof(bool), hipMemcpyHostToDevice));

        //call kernel 1
        computeIntermediatesAndPath<<<blockPerGrid, threadsPerBlock>>>(d_c, vertex, edges, threadMask, cost, intermediateCostAndPath);
        hipDeviceSynchronize();

        //call kernel 2
        computeFinalCosts<<<blockPerGrid, threadsPerBlock>>>(d_done_ptr, vertex, edges, threadMask, cost, intermediateCostAndPath);
        hipDeviceSynchronize();

        //memcpy d_done to h_done
        CHECK(hipMemcpy(&h_done, d_done_ptr, sizeof(bool), hipMemcpyDeviceToHost));
        

    }

    #if(TEST)
        printPath(intermediateCostAndPath);
    #endif

    
    // pathElement x;
    // x.value = 1.2f;
    // x.pathIndex = 5;

    // std::cout << std::bitset<32>(*(reinterpret_cast<int*>(&x.value))) << std::endl;
    // std::cout << std::bitset<32>(x.pathIndex) <<std::endl;

    // unsigned long long int y = __pathElement_as_ulli(&x);
    // std::cout << std::bitset<64>(y) << std::endl;

    // float xval = __ulli_as_pathElement(&y)->value;
    // int xpathIndex = (__ulli_as_pathElement(&y))->pathIndex;

    // std::cout << std::bitset<32>(*(reinterpret_cast<int*>(&xval))) << std::endl;
    // std::cout << std::bitset<32>(xpathIndex) << std::endl;
    
    
    printf("\ncost of target - %f\n", d_c[minElement[0].row * MATRIX_SIZE + minElement[0].col] + cost[minElement[1].row * MATRIX_SIZE + minElement[1].col] - d_c[minElement[1].row * MATRIX_SIZE + minElement[1].col]);       //include source's weight and exclude target's weight
}